#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/JitLoops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/Pow.h>
#include <c10/core/Scalar.h>

namespace at { namespace native {

// Forward declare some unary kernels
void rsqrt_kernel_cuda(TensorIteratorBase& iter);
void sqrt_kernel_cuda(TensorIteratorBase& iter);
void reciprocal_kernel_cuda(TensorIteratorBase& iter);

namespace {


// SFINAE doesn't work well with NVCC under Windows for math functions like pow and sqrt.
// So we need to define the functions with the explicit function signatures.
// As for pow, the following signatures are defined as the device function:
//   pow(float, int)
//   pow(double, int)
//   pow(float, float)
//   pow(double, double)
#ifdef _MSC_VER
// Functions for pow
// pow for at::Half
static inline __host__ __device__ at::Half pow_(at::Half base, at::Half exp) {
  return static_cast<at::Half>(std::pow(static_cast<float>(base), static_cast<float>(exp)));
}
// pow for at::BFloat16
static inline __host__ __device__ at::BFloat16 pow_(at::BFloat16 base, at::BFloat16 exp) {
  return static_cast<at::BFloat16>(std::pow(static_cast<float>(base), static_cast<float>(exp)));
}
// pow (floating, floating/int)
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ typename std::enable_if<std::is_floating_point<Base_type>::value && (std::is_same<Base_type, Exp_type>::value || std::is_same<Exp_type, int>::value), Base_type>::type
  pow_(Base_type base, Exp_type exp) {
  return std::pow(base, exp);
}
// pow (Otherwise)
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ typename std::enable_if<!std::is_same<Base_type, Exp_type>::value && !std::is_same<Exp_type, int>::value, Base_type>::type
  pow_(Base_type base, Exp_type exp) {
  return static_cast<Base_type>(std::pow(static_cast<double>(base), static_cast<double>(exp)));
}
#else
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ Base_type pow_(Base_type base, Exp_type exp) {
  return ::pow(base, exp);
}
#endif

template <typename T>
static inline __host__ __device__ std::enable_if_t<std::is_integral<T>::value, T> pow_(
    T base, T exp) {
  return at::native::powi(base, exp);
}

template <typename T>
static inline __host__ __device__ c10::complex<T> pow_(c10::complex<T> base, c10::complex<T> exp) {
  return c10_complex_math::pow(base, exp);
}

void pow_tensor_scalar_kernel(TensorIteratorBase& iter, const Scalar& exp_scalar);

template <typename scalar_t>
void pow_scalar_tensor_impl(TensorIteratorBase& iter, scalar_t base) {
  gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t exp) -> scalar_t {
    return pow_(base, exp);
  });
}

template <typename value_t>
void pow_scalar_tensor_impl(TensorIteratorBase& iter, c10::complex<value_t> base) {
  // For complex, thrust::pow uses the identity
  // pow(a, b) = exp(log(a) * b)
  const auto fct = std::log(base);
  gpu_kernel(iter, [=]GPU_LAMBDA(c10::complex<value_t> exp) -> c10::complex<value_t> {
    return std::exp(fct * exp);
  });
}

/* complex<Half> support impl */
const char pow_scalar_base_name[] = "pow_scalar_base_kernel";
template <>
void pow_scalar_tensor_impl(TensorIteratorBase& iter, c10::complex<at::Half> base) {
  using scalar_t = c10::complex<at::Half>;
  using opmath_t = at::opmath_type<scalar_t>;
  // For complex, thrust::pow uses the identity
  // pow(a, b) = exp(log(a) * b)
  const auto fct = std::log(opmath_t{base});
#if AT_USE_JITERATOR()
  static const auto pow_kernel_string =
      jiterator_stringify(template <typename T> T pow_scalar_base_kernel(T exp, T fct) {
        return std::exp(fct * exp);
      });
  jitted_gpu_kernel<pow_scalar_base_name, scalar_t, scalar_t, 1>(
      iter,
      pow_kernel_string,
      /*scalar_pos=*/at::cuda::jit::BinaryFuncVariant::NoScalar,
      /*scalar_val=*/0,
      /*extra_args=*/std::make_tuple(fct));
#else
  gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t exp) -> scalar_t {
    return std::exp(fct * opmath_t{exp});
  });
#endif
}

namespace {

#if AT_USE_JITERATOR()
/* complex<Half> support impl */
const char pow_name[] = "pow_kernel";
static const auto pow_kernel_string =
    jiterator_stringify(template <typename T> T pow_kernel(T base, T exp) {
      return std::pow(base, exp);
    });
#endif

/* complex<Half> support impl */
void pow_chalf_tensor_scalar_impl(TensorIteratorBase& iter, const Scalar& exp_scalar) {
  using scalar_t = c10::complex<at::Half>;
  using opmath_t = at::opmath_type<scalar_t>;
  auto exp = exp_scalar.to<opmath_t>();
#if AT_USE_JITERATOR()
  jitted_gpu_kernel<pow_name, scalar_t, scalar_t, 1>(
      iter,
      pow_kernel_string,
      /*scalar_pos=*/at::cuda::jit::BinaryFuncVariant::NoScalar,
      /*scalar_val=*/0,
      /*extra_args=*/std::make_tuple(exp));
#else
  gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t base) -> scalar_t {
    return std::pow(opmath_t{base}, exp);
  });
#endif
}

}  // anonymous namespace

void pow_tensor_tensor_kernel(TensorIteratorBase& iter) {
  auto common_dtype = iter.common_dtype();
  if (common_dtype == kComplexHalf) {
    using scalar_t = c10::complex<at::Half>;
    if (iter.is_cpu_scalar(1)) {
      const auto base = iter.scalar_value<scalar_t>(1);
      iter.remove_operand(1);
      pow_scalar_tensor_impl(iter, base);
    } else if (iter.is_cpu_scalar(2)) {
      const auto exp = iter.scalar_value<scalar_t>(2);
      iter.remove_operand(2);
      pow_chalf_tensor_scalar_impl(iter, exp);
    } else {
      using opmath_t = at::opmath_type<scalar_t>;
      TORCH_INTERNAL_ASSERT(!iter.is_cpu_scalar(1) && !iter.is_cpu_scalar(2));
#if AT_USE_JITERATOR()
      jitted_gpu_kernel<pow_name, scalar_t, scalar_t, 2>(
          iter, pow_kernel_string);
#else
      gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
            using opmath_t = at::opmath_type<scalar_t>;
            return pow_(opmath_t{base}, opmath_t{exp});
          });
#endif
    }
  } else {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
        kHalf, kBFloat16, iter.common_dtype(), "pow_cuda", [&] {
      if (iter.is_cpu_scalar(1)) {
        const auto base = iter.scalar_value<scalar_t>(1);
        iter.remove_operand(1);
        pow_scalar_tensor_impl(iter, base);
      } else if (iter.is_cpu_scalar(2)) {
        const auto exp = iter.scalar_value<scalar_t>(2);
        iter.remove_operand(2);
        pow_tensor_scalar_kernel(iter, exp);
      } else {
        gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
          return pow_(base, exp);
        });
      }
    });
  }
}


template<typename Base_type, typename Exp_type>
void pow_tensor_scalar_kernel_impl(TensorIteratorBase& iter,
                                                 Exp_type exp) {
  const auto d_exp = static_cast<double>(exp);
  // .5 (sqrt), -.5 (rsqrt) and -1 (reciprocal) specializations are handled
  // in pow_tensor_scalar_kernel
  if (d_exp == 2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base;
    });
  } else if (d_exp == 3) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base * base;
    });
  } else if (d_exp == -2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return 1.0 / (base * base);
    });
  } else {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return pow_(base, exp);
    });
  }
}

void pow_tensor_scalar_kernel(TensorIteratorBase& iter, const Scalar& exp_scalar) {
  // Dispatch to fast specialization for sqrt, rsqrt and reciprocal
  if (!exp_scalar.isComplex()) {
    if (exp_scalar.equal(.5)) {
      return sqrt_kernel_cuda(iter);
    } else if (exp_scalar.equal(-0.5)) {
      return rsqrt_kernel_cuda(iter);
    } else if (exp_scalar.equal(-1.0)) {
      return reciprocal_kernel_cuda(iter);
    }
  }
  if (isComplexType(iter.common_dtype()) || exp_scalar.isComplex()) {
    if (iter.common_dtype() == kComplexHalf) {
      using scalar_t = c10::complex<at::Half>;
      pow_chalf_tensor_scalar_impl(iter, exp_scalar);
      return;
    }
    AT_DISPATCH_COMPLEX_TYPES(iter.common_dtype(), "pow_cuda", [&]() {
      const auto exp = exp_scalar.to<scalar_t>();
      gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t base) -> scalar_t {
        return pow_(base, exp);
      });
    });
  } else if (isFloatingType(iter.common_dtype()) || exp_scalar.isIntegral(false)) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "pow_cuda", [&]() {
      const auto exp = exp_scalar.to<scalar_t>();
      pow_tensor_scalar_kernel_impl<scalar_t>(iter, exp);
    });
  } else {
    TORCH_INTERNAL_ASSERT(false, "invalid combination of type in Pow function, common dtype:", iter.common_dtype(),
                                 "exp is integral?", exp_scalar.isIntegral(false));
  }
}

} // anonymous namespace

REGISTER_DISPATCH(pow_tensor_tensor_stub, &pow_tensor_tensor_kernel);
REGISTER_DISPATCH(pow_tensor_scalar_stub, &pow_tensor_scalar_kernel);

}} // namespace at::native
